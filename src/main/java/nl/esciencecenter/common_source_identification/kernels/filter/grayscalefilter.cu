
#include <hip/hip_runtime.h>
/*
* Copyright 2015 Netherlands eScience Center, VU University Amsterdam, and Netherlands Forensic Institute
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
* http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#ifndef block_size_x
#define block_size_x 32
#endif

#ifndef block_size_y
#define block_size_y 16
#endif

/**
 * This file contains the CUDA kernel for converting an image into
 * a grayscale array of floats. Scaling factors used are:
 * 0.299 r + 0.587 g + 0.114 b
 *
 * @author Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
 * @version 0.1
 */
extern "C" {
//    __global__ void grayscale(int h, int w, float* output, uchar3* input);
    __global__ void grayscale(int h, int w, float* output, char* input);
}

/*
 * Naive grayscale kernel
 *
 * Bytes go in, floats come out, alpha is ignored
 *
 * gridDim.x = w / block_size_x  (ceiled)
 * gridDim.y = h / block_size_y  (ceiled)
 */
//__global__ void grayscale(int h, int w, float* output, uchar3* input) {
__global__ void grayscale(int h, int w, float* output, char* input) {
	int i = threadIdx.y + blockIdx.y * block_size_y;
	int j = threadIdx.x + blockIdx.x * block_size_x;
	
    uchar3 *c3_input = (uchar3 *)input;

	if (j < w && i < h) {

		uchar3 c = c3_input[i*w+j];

//          float b = (float) input[(i*w+j) * 3 + 0] & 0xFFFF;
//          float g = (float) input[(i*w+j) * 3 + 1] & 0xFFFF;
//          float r = (float) input[(i*w+j) * 3 + 2] & 0xFFFF;
		output[i*w+j] = 0.299f*c.z + 0.587f*c.y + 0.114f*c.x;




	}
}



