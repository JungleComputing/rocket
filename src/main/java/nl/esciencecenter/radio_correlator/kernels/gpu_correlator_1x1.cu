#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Rob van Nieuwpoort & John Romein
Astron
P.O.Box 2, 7990 AA Dwingeloo, The Netherlands, nieuwpoort@astron.nl

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/
#include <hipcub/hipcub.hpp>
#include <hip/hip_complex.h>

struct float8 {
  float a;
  float b;
  float c;
  float d;
  float e;
  float f;
  float g;
  float h;
};

__forceinline__ __host__ __device__ float8 make_float8(float a, float b, float c, float d, float e, float f, float g, float h) {
  float8 output = { a, b, c, d, e, f, g, h };
  return output;
}

__forceinline__ __host__ __device__ float8 operator+(float8 x, float8 y) {
    return make_float8(x.a+y.a, x.b+y.b, x.c+y.c, x.d+y.d, x.e+y.e, x.f+y.f, x.g+y.g, x.h+y.h);
}

extern "C"
__global__ void correlate(
        unsigned numTimes,
        const float4 *__restrict__ left,
        const float4 *__restrict__ right,
        float2 *__restrict__ result
) {
    typedef hipcub::BlockReduce<float8, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    unsigned myBlock   = blockIdx.x;
    unsigned myThread  = threadIdx.x;

    hipFloatComplex xx = make_hipFloatComplex(0.0, 0.0);
    hipFloatComplex xy = make_hipFloatComplex(0.0, 0.0);
    hipFloatComplex yx = make_hipFloatComplex(0.0, 0.0);
    hipFloatComplex yy = make_hipFloatComplex(0.0, 0.0);

    for (unsigned t = myThread; t < numTimes; t += BLOCK_SIZE) {
		float4 sample0 = left[numTimes * myBlock + t];
		float4 sample1 = right[numTimes * myBlock + t];

		hipFloatComplex lx = make_hipFloatComplex(sample0.x, sample0.y);
		hipFloatComplex ly = make_hipFloatComplex(sample0.z, sample0.w);
		hipFloatComplex rx = make_hipFloatComplex(sample1.x, sample1.y);
		hipFloatComplex ry = make_hipFloatComplex(sample1.z, sample1.w);

        xx = hipCaddf(xx, hipCmulf(lx, hipConjf(rx)));
        xy = hipCaddf(xy, hipCmulf(lx, hipConjf(ry)));
        yx = hipCaddf(yx, hipCmulf(ly, hipConjf(rx)));
        yy = hipCaddf(yy, hipCmulf(ly, hipConjf(ry)));
    }

    float8 out = BlockReduce(temp_storage).Sum(make_float8(
        hipCrealf(xx), hipCimagf(xx),
        hipCrealf(xy), hipCimagf(xy),
        hipCrealf(yx), hipCimagf(yx),
        hipCrealf(yy), hipCimagf(yy)
    ));

    if (myThread == 0) {
        result[4 * myBlock + 0] = make_hipFloatComplex(out.a, out.b);
        result[4 * myBlock + 1] = make_hipFloatComplex(out.c, out.d);
        result[4 * myBlock + 2] = make_hipFloatComplex(out.e, out.f);
        result[4 * myBlock + 3] = make_hipFloatComplex(out.g, out.h);
    }
}
